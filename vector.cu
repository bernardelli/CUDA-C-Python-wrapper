#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <>
#include <tiny_helper_cuda.h>

__global__ void
vectormult_kernel(float *A, float k, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		A[i] *= k;
	}
}




extern "C"
void vectormult(float* dev_vector, float k, int size)
{
	const int n_threads = 128;
	const int n_blocks = (size + n_threads - 1) / n_threads;
	vectormult_kernel << <n_blocks, n_threads >> >(dev_vector, k, size);
	checkCudaErrors(hipDeviceSynchronize());
	getLastCudaError("vectormult_kernel");
}